#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/param.h>
#include "utils.h"

typedef unsigned char guint8;


__global__ void zcartoon_kernel(
        guint8 *data,
        int width,
        int height,
        int top,
        int m_mask_radius,
        float m_threshold,
        float m_ramp,
        float m_size
        )
    {

    int x = ((blockIdx.x * blockDim.x) + threadIdx.x);
    int y = ((blockIdx.y * blockDim.y) + threadIdx.y);

    int m_pixelPos = (y * width + x) * 4; // main pixel

    // get neighbour pixels
    int i = 0;
    int sumR = 0, sumB = 0, sumG = 0;
    for(int iX = x-top; i < m_mask_radius; ++i, ++iX){

      int j = 0;
      for(int iY = y-top; j < m_mask_radius; ++j, ++iY){

        int n_pixelPos = (iY * width + iX) * 4; // neighbour pixel
        sumR += data[n_pixelPos + 2];
        sumB += data[n_pixelPos + 0];
        sumG += data[n_pixelPos + 1];
      }
    }

    float red = data[m_pixelPos + 2],
           blue = data[m_pixelPos + 0],
           green = data[m_pixelPos + 1];

    float koeffR = red / (sumR / m_size),
           koeffB = blue / (sumB / m_size),
           koeffG = green / (sumG / m_size);

    if(koeffR < m_threshold)
        red *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffR)))/m_ramp);

    if(koeffB < m_threshold)
        blue *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffB)))/m_ramp);

    if(koeffG < m_threshold)
        green *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffG)))/m_ramp);

    data[m_pixelPos + 2] = red;
    data[m_pixelPos + 0] = blue;
    data[m_pixelPos + 1] = green;

}

void zcartoon_transform( guint8 *data, int width, int height ){
    guint8 *d_data;
    int m_mask_radius = 7;
    float m_threshold = 1.0;
    float m_ramp = 0.1;
    size_t size = width * height * 4;

    float m_size = m_mask_radius * m_mask_radius;
    int top = m_mask_radius / 2;

    checkCudaErrors( hipMalloc( (void**)&d_data, size ) );
    checkCudaErrors( hipMemcpy( d_data, data, size, hipMemcpyHostToDevice ) );

    dim3 threads = dim3(32, 32);
    dim3 blocks = dim3(width / threads.x, height / threads.y);
    //printf("Threads x: %d, y: %d; blocks x: %d, y: %d\n", threads.x, threads.y, blocks.x, blocks.y);

    // execute kernel
    zcartoon_kernel<<< blocks, threads >>>( d_data, width, height, top, m_mask_radius, m_threshold, m_ramp, m_size );

    checkCudaErrors( hipMemcpy( data, d_data, size, hipMemcpyDeviceToHost ) );

    hipFree( d_data );
}
