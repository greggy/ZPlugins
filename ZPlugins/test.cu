#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/param.h>
#include "utils.h"

typedef unsigned char guint8;


__global__ void transform(
        guint8 *data,
        guint8 *o_data,
        int width,
        int top,
        int m_mask_radius,
        double m_threshold,
        double m_ramp,
        double m_size
        )
    {
    uint x = ((blockIdx.x * blockDim.x) + threadIdx.x);
    uint y = ((blockIdx.y * blockDim.y) + threadIdx.y);

    uint m_pixelPos = (y * width + x) * 4; // main pixel

    // get neighbour pixels
    int i = 0;
    double sumR = 0, sumB = 0, sumG = 0;
    for(int iX = x-top; i < m_mask_radius; ++i, ++iX){

      int j = 0;
      for(int iY = y-top; j < m_mask_radius; ++j, ++iY){

        uint n_pixelPos = (iY * width + iX) * 4; // neighbour pixel
        sumR += o_data[n_pixelPos + 2];
        sumB += o_data[n_pixelPos + 0];
        sumG += o_data[n_pixelPos + 1];
      }
    }

    sumR /= m_size;
    sumB /= m_size;
    sumG /= m_size;

    double red = o_data[m_pixelPos + 2],
           blue = o_data[m_pixelPos + 0],
           green = o_data[m_pixelPos + 1];

    double koeffR = red / sumR,
           koeffB = blue / sumB,
           koeffG = green / sumG;

    if(koeffR < m_threshold)
        red *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffR)))/m_ramp);

    if(koeffB < m_threshold)
        blue *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffB)))/m_ramp);

    if(koeffG < m_threshold)
        green *= ((m_ramp - MIN(m_ramp,(m_threshold - koeffG)))/m_ramp);

    data[m_pixelPos + 2] = red;
    data[m_pixelPos + 0] = blue;
    data[m_pixelPos + 1] = green;

}

void zcartoon_transform( guint8 *data, guint8 *o_data, int i_width, int i_height ){
    guint8 *d_data;
    guint8 *do_data;
    int m_mask_radius = 7;
    double m_threshold = 1.0;
    double m_ramp = 0.1;
    size_t size = i_width * i_height * 4;

    //printf ("Image length %d", len);

    //int center = m_mask_radius / 2 + 1,
            //width = i_width - center,
            //height = i_height - center,
    int top = m_mask_radius / 2;


    checkCudaErrors( hipMalloc( (void**)&d_data, size ) );
    checkCudaErrors( hipMalloc( (void**)&do_data, size ) );
    checkCudaErrors( hipMemcpy( d_data, data, size, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( do_data, data, size, hipMemcpyHostToDevice ) );

    dim3 threads = dim3(8, 8);
    dim3 blocks = dim3(i_width / threads.x, i_height / threads.y);

    double m_size = m_mask_radius * m_mask_radius;

    // execute kernel
    transform<<< blocks, threads >>>( d_data, do_data, i_width, top, m_mask_radius, m_threshold, m_ramp, m_size );

    checkCudaErrors( hipMemcpy( data, d_data, size, hipMemcpyDeviceToHost ) );

//    for (int i = 0; i < width * height; i++){
//        int b = data[i + 0];
//        int g = data[i + 1];
//        int r = data[i + 2];
//        if (b > 0 || g > 0 || r > 0)
//            printf ("%d, %d, %d\n", b, g, r);
//    }

    hipFree( d_data );
    hipFree( do_data );
    free( o_data );
}


// test kernel
__global__ void add( int a, int b, int *c){
           *c = a + b;
}

int test( int len ){
    int c;
    int *dev_c;

    checkCudaErrors( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1, 1>>>( 2, 7, dev_c );

    checkCudaErrors( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost ) );

    printf( "2 + 7 = %d and %d\n", c, len );
    hipFree( dev_c );

    return 0;
}
