#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/param.h>
#include "utils.h"

typedef unsigned char guint8;


__global__ void simple_kernel( guint8 *data, int width )
    {
    uint x = ((blockIdx.x * blockDim.x) + threadIdx.x);
    uint y = ((blockIdx.y * blockDim.y) + threadIdx.y);

    uint pixelPos = (y * width + x) * 4;
    data[pixelPos + 2] = 0;
}

void simple_transform( guint8 *data, int i_width, int i_height ){
    guint8 *d_data;
    size_t size = i_width * i_height * 4;

    //printf ("Image length %d", len);


    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    checkCudaErrors( hipMalloc( (void**)&d_data, size ) );
    checkCudaErrors( hipMemcpy( d_data, data, size, hipMemcpyHostToDevice ) );

    dim3 threads = dim3(8, 8);
    dim3 blocks = dim3(i_width / threads.x, i_height / threads.y);

    // execute kernel
    simple_kernel<<< blocks, threads >>>( d_data, i_width );

    checkCudaErrors( hipMemcpy( data, d_data, size, hipMemcpyDeviceToHost ) );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
//    printf("Frame was proccessed during: %f\n", elapsedTime);

    hipFree( d_data );
}


// test kernel
__global__ void add( int a, int b, int *c){
           *c = a + b;
}

int test( int len ){
    int c;
    int *dev_c;

    checkCudaErrors( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1, 1>>>( 2, 7, dev_c );

    checkCudaErrors( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost ) );

    printf( "2 + 7 = %d and %d\n", c, len );
    hipFree( dev_c );

    return 0;
}
