#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

typedef unsigned char guint8;


__constant__ float cGaussian[64];   //gaussian array in device side
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;

uint *dImage  = NULL;   //original image
size_t pitch;


//Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
__device__ float euclideanLen(float4 a, float4 b, float d)
{

    float mod = (b.x - a.x) * (b.x - a.x) +
                (b.y - a.y) * (b.y - a.y) +
                (b.z - a.z) * (b.z - a.z);

    return __expf(-mod / (2.f * d * d));
}

__device__ uint rgbaFloatToInt2(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));
    return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}

__device__ float4 rgbaIntToFloat(uint c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
    return rgba;
}

//column pass using coalesced global memory reads
__global__ void
bilateral_transform(guint8 *od, int w, int h,
                    float e_d,  int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center = tex2D(rgbaTex, x, y);

    for (int i = -r; i <= r; i++)
    {
        for (int j = -r; j <= r; j++)
        {
            float4 curPix = tex2D(rgbaTex, x + j, y + i);
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen(curPix, center, e_d);             //range factor

            t += factor * curPix;
            sum += factor;
        }
    }

    od[y * w + x] = rgbaFloatToInt2(t/sum);
}


void updateGaussian(float delta, int radius)
{
    float  fGaussian[64];

    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i-radius;
        fGaussian[i] = expf(-(x*x) / (2*delta*delta));
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1)));
}


void bilateral_transform( guint8 *data, int width, int height ){
    //guint8 *d_data;
    float gaussian_delta = 4;
    float e_d = 0.1f;
    int radius = 5;
    //size_t size = width * height * 4;

    updateGaussian(gaussian_delta, radius);

    // copy image data to array
    checkCudaErrors(hipMallocPitch(&dImage, &pitch, sizeof(guint8) * width, height));
    checkCudaErrors(hipMemcpy2D(dImage, pitch, data, sizeof(guint8) * width,
                                 sizeof(guint8) * width, height, hipMemcpyHostToDevice));
    printf("Pitch: %d\n", pitch);

    // bind array to texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    checkCudaErrors(hipBindTexture2D(0, rgbaTex, dImage, desc, width, height, pitch));

    dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
    dim3 blockSize(16, 16);
    bilateral_transform<<<gridSize, blockSize>>>(data, width, height, e_d, radius);

    checkCudaErrors(hipFree(dImage));
    checkCudaErrors(hipUnbindTexture(rgbaTex));

//    // copy image data to array
//    hipChannelFormatDesc desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
//    checkCudaErrors(hipMallocArray(&dImage, &desc, width, height));
//    checkCudaErrors(hipMalloc(&d_data, size));
//    checkCudaErrors(hipMemcpyToArray(dImage, 0, 0, data, size, hipMemcpyHostToDevice));

//    // Bind the array to the texture
//    checkCudaErrors(hipBindTextureToArray(rgbaTex, dImage));

//    dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
//    dim3 blockSize(16, 16);
//    bilateral_transform<<< gridSize, blockSize >>>(d_data, width, height, e_d, radius);

//    checkCudaErrors(hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost));

//    checkCudaErrors(hipFreeArray(dImage));
//    checkCudaErrors(hipFree(d_data));
//    checkCudaErrors(hipUnbindTexture(rgbaTex));

}
